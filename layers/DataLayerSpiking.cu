#include "hip/hip_runtime.h"
#include "DataLayerSpiking.h"
#include "opencv2/opencv.hpp"
#include <vector>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
//#include <thread>
#include "../common/Config.h"
#include "../common/cuBase.h"
#include "../common/util.h"
#include "../dataAugmentation/cuTransformation.cuh"

#define CONST_SPIKING_SCALE (5.5 * 255.0f)

hiprandGenerator_t rand_gen_device;
const hiprandRngType_t gen_t = HIPRAND_RNG_PSEUDO_DEFAULT;

/*
 * dim3 block = dim3(batch, outputAmount);
 * dim3 thread= dim3(min(outputDim * endTime, 1024));
*/
__global__ void g_dataLayer_spiking_feedforward(
	bool** inputs,
	bool* outputs,
    int outputArea,
    int outputCols);

DataLayerSpiking::DataLayerSpiking(std::string name){
	m_name = name;
    myId = 0;

    ConfigDataSpiking* config = (ConfigDataSpiking*)Config::instance()->getLayerByName(m_name);
	inputDim  = config->m_inputNeurons;
	outputDim = inputDim;
    endTime   = Config::instance()->getEndTime();
	batch     = Config::instance()->getBatchSize();
    imgSize   = Config::instance()->getImageSize();
	inputAmount = Config::instance()->getChannels();
	outputAmount= inputAmount;
	outputs = new cuMatrix<bool>(batch, endTime * outputDim, outputAmount);
    outputs_time = new cuMatrix<int>(batch, outputDim * endTime, outputAmount);

    fireCount = new cuMatrix<int>(batch, outputDim, outputAmount);

    cu_randomNum = new cuMatrix<float>(batch, endTime * inputDim, 1);

    bool has_distortion = Config::instance()->applyPreproc();
    for(int i = 0; i < 2; ++i){
        for(int j = 0; j < batch; j++){
            batchSpeeches[i].push_back(new cuMatrix<bool>(endTime, inputDim, Config::instance()->getChannels()));
            if(has_distortion){
                batchSamplesFloat[i].push_back(new cuMatrix<float>(imgSize, imgSize, Config::instance()->getChannels()));
            }
        }
        batchSpeeches[i].toGpu();
        if(has_distortion)
            batchSamplesFloat[i].toGpu();
    }
    if(has_distortion){
        for(int i = 0; i < batch; ++i){
            processOutputs.push_back(new cuMatrix<float>(imgSize, imgSize, Config::instance()->getChannels()));
        }
        processOutputs.toGpu();
    }

	checkCudaErrors(hipStreamCreate(&stream1));

	Layers::instance()->set(m_name, this);

	hiprandStatus_t curandstatus = hiprandCreateGenerator(&rand_gen_device, gen_t);
	if(curandstatus != HIPRAND_STATUS_SUCCESS)
	{
		char logStr[1024];
		sprintf(logStr, "DataLayerSpiking::hiprandCreateGenerator fail\n");
		LOG(logStr, "Result/log.txt");
		assert(0);
	}
    
}


/*
 * dim3 block = dim3(batch, inputDim);
 * dim3 thread= dim3(min(1024, endTime));
*/
__global__ void g_dataLayer_poissonCode(
    float** preprocs,
    bool** inputs,
    float* _randoms,
    int batch,
    int inputDim,
    int endTime)
{
    int batchId = blockIdx.x;
    int i_idx = blockIdx.y;
    int speechSize = endTime * inputDim;

    float * random = _randoms + batchId * speechSize;
    float * preproc = preprocs[batchId];
    bool * input = inputs[batchId];
    float distorted = preproc[i_idx];
    float freq = ((distorted + 1) * 255.0f / 2) / CONST_SPIKING_SCALE; // map back to freq range;
    for(int t = 1; t < endTime; t += blockDim.x)
    {
        int time = t + threadIdx.x;
        float r = random[time * inputDim + i_idx];
        if(r < freq)    input[time * inputDim + i_idx] = true;
        else    input[time * inputDim + i_idx] = false;
    }

}

/*
 * dim3 block = dim3(batch, outputAmount);
 * dim3 thread= dim3(min(outputDim * endTime, 1024));
*/

__global__ void g_dataLayer_spiking_feedforward(
	bool** inputs,
	bool* outputs,
    int outputArea,
    int outputCols)
{
	int batchId = blockIdx.x;
    int ok      = blockIdx.y;

    int outputAmount = gridDim.y;

	bool* input  = inputs[batchId];
	bool* output = outputs + ok * outputArea+ batchId * outputCols * outputAmount;
	for(int i = 0; i < outputCols; i += blockDim.x){
		int idx = i + threadIdx.x;
		if(idx < outputCols){
			output[idx] = input[idx];
		}
	}
}

/*
 * dim3 block = dim3(batch);
 * dim3 thread= dim3(min(outputDim, 1024));
*/
__global__ void g_dataLayer_get_fireCount(
    bool* outputs,
    int* batchfireCount,
    int outputDim,
    int endTime)
{
	int batchId = blockIdx.x;

    bool* output = outputs + batchId * endTime * outputDim;
    int* fireCount = batchfireCount + batchId * outputDim;

    for(int i = 0; i < outputDim; i += blockDim.x)
    {
        int o_idx = i + threadIdx.x;
        if(o_idx < outputDim){
            int sum = 0;
            for(int time = 0; time < endTime; ++time)   sum += output[o_idx + time * outputDim];
            fireCount[o_idx] = sum;
        }
    }
}


//* simply copy the input data to the output
void DataLayerSpiking::feedforward(){
	dim3 block = dim3(batch, outputAmount);
	dim3 thread= dim3(min(outputDim * endTime, 1024));
	
	g_dataLayer_spiking_feedforward<<<block, thread>>>(
		batchSpeeches[myId].m_devPoint, 
		outputs->getDev(),
		outputs->getArea(),
        outputs->cols);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("DataLayerSpiking:feedforward");

    //* get the fire counts for transforming the binary response to spike times    
    thread = dim3(min(outputDim, 1024));
    g_dataLayer_get_fireCount<<<block, thread>>>(
        outputs->getDev(),
        fireCount->getDev(),
        outputDim,
        endTime);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("DataLayerSpiking:g_dataLayer_get_fireCount");
    

    g_response_2_spiketime<<<block, thread>>>(
        outputs->getDev(),
        outputs_time->getDev(),
        outputs->getArea(),
        outputDim,
        endTime);
    checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("DataLayerSpiking:g_response_2_spiketime");

}; 

//* apply the distortation here
void DataLayerSpiking::getBatchSpikesWithPreproc(cuMatrixVector<bool>& inputs, int start)
{
    assert(Config::instance()->applyPreproc() == true);
    
    // generate the uniform random number
    generateRandom(clock() + start);

    // cp the float raw sample to Gpu
    int id = 1 - this->myId;
    for(size_t i = 0; i < this->batchSamplesFloat[id].size(); i++){
        memcpy(this->batchSamplesFloat[id][i]->getHost(), inputs[i + start]->getHostRawImg(), sizeof(float) * this->batchSamplesFloat[id][i]->getLen());
        this->batchSamplesFloat[id][i]->toGpu(this->stream1);
        this->batchSpeeches[id][i]->toGpu(this->stream1);
    }
    // apply the distortation
    cuApplyDistortion(batchSamplesFloat[id].m_devPoint, processOutputs.m_devPoint, batch, imgSize); 

    // map the distorted samples to spike times
    g_dataLayer_poissonCode<<<dim3(batch, inputDim), dim3(min(1024, endTime))>>>(processOutputs.m_devPoint, batchSpeeches[id].m_devPoint, cu_randomNum->getDev(), batch, inputDim, endTime);
    /* // do the same thing by CPU
    for(size_t i = 0; i < this->processOutputs.size(); i++){
        processOutputs[i]->toCpu();
        convertToSpikeTimes(processOutputs[i], inputs[i+start]->getSpikeTimes(), imgSize, endTime);
    }
    */
    // show the distorted image
    if (Config::instance()->getImageShow()) {
		for (int ff = batch - 1; ff >= 0; ff--) {
			showImg(batchSamplesFloat[id][ff], 5);
            processOutputs[ff]->toCpu();
			showImg(processOutputs.m_vec[ff], 5);
			cv::waitKey(0);
		}
	}
}

void DataLayerSpiking::testData()
{
}

//* generate the random numbers for map preproc samples to poisson spike trains
void DataLayerSpiking::generateRandom(unsigned long long seed)
{
	hiprandGenerateUniform(rand_gen_device, cu_randomNum->getDev(), cu_randomNum->getLen());
}

void DataLayerSpiking::synchronize(){
    myId = 1 - myId;
    hipStreamSynchronize(this->stream1);
}

//* get the input spike trains in batch from the input speeches streams
void DataLayerSpiking::getBatchSpikes(cuMatrixVector<bool>& inputs, int start){
    int id = 1 - this->myId;
    for(size_t i = 0; i < this->batchSpeeches[id].size(); i++){
        inputs[i+start]->sparseToDense();
        memcpy(this->batchSpeeches[id][i]->getHost(), inputs[i + start]->getHost(), sizeof(bool) * this->batchSpeeches[id][i]->getLen());
        this->batchSpeeches[id][i]->toGpu(this->stream1);
        inputs[i+start]->freeCpuMem();
        //this->batchSpeeches[i]->toGpu();
    }
}


void DataLayerSpiking::loadBatchSpikes(cuMatrixVector<bool>& inputs, int start){
    if(Config::instance()->applyPreproc() == true)
        getBatchSpikesWithPreproc(inputs, start);
    else
        getBatchSpikes(inputs, start);
}
