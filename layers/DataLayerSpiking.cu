#include "hip/hip_runtime.h"
#include "DataLayerSpiking.h"
#include "opencv2/opencv.hpp"
#include <vector>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
//#include <thread>
#include "../common/Config.h"
#include "../common/cuBase.h"
#include "../common/util.h"


/*
 * dim3 block = dim3(batch, outputAmount);
 * dim3 thread= dim3(min(outputDim * endTime, 1024));
*/
__global__ void g_dataLayer_spiking_feedforward(
	bool** inputs,
	bool* outputs,
    int outputArea,
    int outputCols);

DataLayerSpiking::DataLayerSpiking(std::string name){
	m_name = name;
    myId = 0;

    ConfigDataSpiking* config = (ConfigDataSpiking*)Config::instance()->getLayerByName(m_name);
	inputDim  = config->m_inputNeurons;
	outputDim = inputDim;
    endTime   = Config::instance()->getEndTime();
	batch     = Config::instance()->getBatchSize();
	inputAmount = Config::instance()->getChannels();
	outputAmount= inputAmount;
	outputs = new cuMatrix<bool>(batch, outputDim * endTime, outputAmount);
    outputs_time = new cuMatrix<int>(batch, outputDim * endTime, outputAmount);

    fireCount = new cuMatrix<int>(batch, outputDim, outputAmount);

    for(int i = 0; i < 2; ++i){
        for(int j = 0; j < batch; j++){
            batchSpeeches[i].push_back(new cuMatrix<bool>(endTime, inputDim, Config::instance()->getChannels()));
        }
        batchSpeeches[i].toGpu();
    }

	checkCudaErrors(hipStreamCreate(&stream1));

	Layers::instance()->set(m_name, this);
}

/*
 * dim3 block = dim3(batch, outputAmount);
 * dim3 thread= dim3(min(outputDim * endTime, 1024));
*/

__global__ void g_dataLayer_spiking_feedforward(
	bool** inputs,
	bool* outputs,
    int outputArea,
    int outputCols)
{
	int batchId = blockIdx.x;
    int ok      = blockIdx.y;

    int outputAmount = gridDim.y;

	bool* input  = inputs[batchId];
	bool* output = outputs + ok * outputArea+ batchId * outputCols * outputAmount;
	for(int i = 0; i < outputCols; i += blockDim.x){
		int idx = i + threadIdx.x;
		if(idx < outputCols){
			output[idx] = input[idx];
		}
	}
}

/*
 * dim3 block = dim3(batch);
 * dim3 thread= dim3(min(outputDim, 1024));
*/
__global__ void g_dataLayer_get_fireCount(
    bool* outputs,
    int* batchfireCount,
    int outputDim,
    int endTime)
{
	int batchId = blockIdx.x;

    bool* output = outputs + batchId * endTime * outputDim;
    int* fireCount = batchfireCount + batchId * outputDim;

    for(int i = 0; i < outputDim; i += blockDim.x)
    {
        int o_idx = i + threadIdx.x;
        int sum = 0;
        for(int time = 0; time < endTime; ++time)   sum += output[o_idx + time * outputDim];
        fireCount[o_idx] = sum;
    }
}


//* simply copy the input data to the output
void DataLayerSpiking::feedforward(){
	dim3 block = dim3(batch, outputAmount);
	dim3 thread= dim3(min(outputDim * endTime, 1024));
	
	g_dataLayer_spiking_feedforward<<<block, thread>>>(
		batchSpeeches[myId].m_devPoint, 
		outputs->getDev(),
		outputs->getArea(),
        outputs->cols);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("DataLayerSpiking:feedforward");

    //* get the fire counts for transforming the binary response to spike times    
    thread = dim3(min(outputDim, 1024));
    g_dataLayer_get_fireCount<<<block, thread>>>(
        outputs->getDev(),
        fireCount->getDev(),
        outputDim,
        endTime);
	checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("DataLayerSpiking:g_dataLayer_get_fireCount");
    

    g_response_2_spiketime<<<block, thread>>>(
        outputs->getDev(),
        outputs_time->getDev(),
        outputDim,
        endTime);
    checkCudaErrors(hipStreamSynchronize(0));
	getLastCudaError("DataLayerSpiking:g_response_2_spiketime");

}; 

void DataLayerSpiking::trainData()
{
}

void DataLayerSpiking::testData()
{
}


void DataLayerSpiking::synchronize(){
    myId = 1 - myId;
    hipStreamSynchronize(this->stream1);
}

//* get the input spike trains in batch from the input speeches streams
void DataLayerSpiking::getBatchSpikesWithStreams(cuMatrixVector<bool>& inputs, int start){
    int id = 1 - this->myId;
    for(size_t i = 0; i < this->batchSpeeches[id].size(); i++){
        memcpy(this->batchSpeeches[id][i]->getHost(), inputs[i + start]->getHost(), sizeof(bool) * this->batchSpeeches[id][i]->getLen());
        this->batchSpeeches[id][i]->toGpu(this->stream1);
        //this->batchSpeeches[i]->toGpu();
    }
}
